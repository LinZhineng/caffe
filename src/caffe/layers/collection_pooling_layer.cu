#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/layers/collection_pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxPoolForward(const int size, const Dtype* bottom_data, const Dtype* bottom_splits,
    const int num, const int num_collections, const int channels, const int height,
    const int width, Dtype* top_data, int* mask, Dtype* top_mask) {
    const int nthreads = size * num_collections;
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype maxval = -FLT_MAX;
    int maxidx = -1; //keep maxidx management in cache
    //calculate which collection am I currently on?
    int l = index / size;
    bottom_data += index % size; //get us to the right pixle in the wrong image
    top_data += index % size;
    for(int n = (l == 0 ? 0 : bottom_splits[l-1]); n < bottom_splits[l]; ++n) {
      //n * size : get us to the correct image
      //index % size : correct position in the image
      if(bottom_data[n * size] > maxval) {
        maxidx = n;
        maxval = bottom_data[n * size];
      }
    }
    top_data[l * size] = maxval;
    if (mask) {
      mask += index % size;
      mask[l * size] = maxidx;
    } else {
      top_mask += index % size;
      top_mask[l * size] = maxidx;
    }
  }
}

template <typename Dtype>
__global__ void AvePoolForward(const int size, const Dtype* bottom_data, const Dtype* bottom_splits,
    const int num, const int num_collections, const int channels, const int height,
    const int width, Dtype* top_data) {
    const int nthreads = size * num_collections;
  CUDA_KERNEL_LOOP(index, nthreads) {
    //calculate which collection am I currently on?
    int l = index / size;
    bottom_data += index % size; //get us to the right pixle in the wrong image
    top_data += index % size;
    Dtype aveval = 0;
    for(int n = (l == 0 ? 0 : bottom_splits[l-1]); n < bottom_splits[l]; ++n) {
      aveval += bottom_data[n * size];
    }
    const int pool_size = bottom_splits[l] - (l == 0 ? 0 : bottom_splits[l-1]);
    top_data[l * size] = aveval / pool_size;
  }
}

template <typename Dtype>
__global__ void StoPoolForwardTrain(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* rand_idx, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + kernel_w, width);
    Dtype cumsum = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
      }
    }
    float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_data[h * width + w];
          return;
        }
      }
    }
  }
}


template <typename Dtype>
__global__ void StoPoolForwardTest(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + kernel_w, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    Dtype cumsum = FLT_MIN;
    Dtype cumvalues = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        cumvalues += bottom_data[h * width + w] * bottom_data[h * width + w];
      }
    }
    top_data[index] = cumvalues / cumsum;
  }
}


template <typename Dtype>
void CollectionPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_splits = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  
  //the number of things we will have to pool together
  const int size = height_ * width_ * channels_;
  
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  int* mask = NULL;
  Dtype* top_mask = NULL;
  switch (this->layer_param_.collection_pooling_param().pool()) {
  case CollectionPoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->mutable_gpu_data();
    } else {
      mask = max_idx_.mutable_gpu_data();
    }
    MaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(size * bottom[1]->num()), CAFFE_CUDA_NUM_THREADS>>>(
        size, bottom_data, bottom_splits, bottom[0]->num(), bottom[1]->num(), channels_,
        height_, width_, top_data, mask, top_mask);
    break;
  case CollectionPoolingParameter_PoolMethod_AVE:
    AvePoolForward<Dtype><<<CAFFE_GET_BLOCKS(size * bottom[1]->num()), CAFFE_CUDA_NUM_THREADS>>>(
        size, bottom_data, bottom_splits, bottom[0]->num(), bottom[1]->num(), channels_,
        height_, width_, top_data);
    break;
  case CollectionPoolingParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED; 
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void MaxPoolBackward(const int size, const Dtype* top_diff, 
    const int* mask, const Dtype* top_mask, const Dtype* collection_splits,
    const int num, const int num_collections, const int channels, const int height,
    const int width, Dtype* bottom_diff) {
    const int nthreads = size * num_collections;
  CUDA_KERNEL_LOOP(index, nthreads) {
    //calculate which collection am I currently on?
    int l = index / size;
    bottom_diff += index % size; //get us to the right pixle in the wrong image
    top_diff += index % size;
    //set the bottom diff vector to zero
    for(int n = (l == 0 ? 0 : collection_splits[l-1]); n < collection_splits[l]; ++n) {
      bottom_diff[n * size] = 0;
    }
    if (mask) {
      mask += index % size;
      bottom_diff[mask[l * size] * size] = top_diff[l * size];
    } else {
      top_mask += index % size;
      bottom_diff[static_cast<int>(top_mask[l * size] * size)] = top_diff[l * size];
    }
  }
}

template <typename Dtype>
__global__ void AvePoolBackward(const int size, const Dtype* top_diff, const Dtype* collection_splits,
    const int num, const int num_collections, const int channels, const int height,
    const int width, Dtype* bottom_diff) {
    const int nthreads = size * num_collections;
  CUDA_KERNEL_LOOP(index, nthreads) {
    //calculate which collection am I currently on?
    int l = index / size;
    const int pool_size = collection_splits[l] - (l == 0 ? 0 : collection_splits[l-1]);
    bottom_diff += index % size; //get us to the right pixle in the wrong image
    top_diff += index % size;
    for(int n = (l == 0 ? 0 : collection_splits[l-1]); n < collection_splits[l]; ++n) {
      bottom_diff[n * size] = top_diff[l * size] / pool_size;
    }
  }
}


template <typename Dtype>
__global__ void StoPoolBackward(const int nthreads,
    const Dtype* rand_idx, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int phend = min(h / stride_h + 1, pooled_height);
    int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    rand_idx += (n * channels + c) * pooled_height * pooled_width;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        gradient += top_diff[ph * pooled_width + pw] *
            (index == static_cast<int>(rand_idx[ph * pooled_width + pw]));
      }
    }
    bottom_diff[index] = gradient;
  }
}


template <typename Dtype>
void CollectionPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                 const vector<bool>& propagate_down, 
                                                 const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const int size = height_ * width_ * channels_;
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  const int* mask = NULL;
  const Dtype* top_mask = NULL;
  switch (this->layer_param_.collection_pooling_param().pool()) {
  case CollectionPoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->gpu_data();
    } else {
      mask = max_idx_.gpu_data();
    }
     MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(size * bottom[1]->num()), CAFFE_CUDA_NUM_THREADS>>>(
         size, top_diff, mask, top_mask, bottom[1]->gpu_data(), top[0]->num(), bottom[1]->num(), channels_,
         height_, width_, bottom_diff);
    break;
  case CollectionPoolingParameter_PoolMethod_AVE:
    AvePoolBackward<Dtype><<<CAFFE_GET_BLOCKS(size * bottom[1]->num()), CAFFE_CUDA_NUM_THREADS>>>(
        size, top_diff, bottom[1]->gpu_data(), top[0]->num(), bottom[1]->num(), channels_,
        height_, width_, bottom_diff);
    break;
  case CollectionPoolingParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(CollectionPoolingLayer);

}  // namespace caffe
